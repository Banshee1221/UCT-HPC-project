#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "med_filter.h"
#include <omp.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 32

//Adopted from http://rosettacode.org/wiki/Averages/Median#Quickselect_algorithm 
__host__ __device__ unsigned int quick_select(int k, unsigned int *x, int len)
{
	int left = 0, right = len - 1;
	int pos, i;
	unsigned __int64 pivot;
	unsigned __int64 t;

	while (left < right)
	{
		pivot = x[k];
		t = x[k];
		x[k] = x[right];
		x[right] = t;

		for (i = pos = left; i < right; i++)
		{
			if (x[i] < pivot)
			{
				t = x[i];
				x[i] = x[pos];
				x[pos] = t;
				pos++;
			}
		}

		t = x[right];
		x[right] = x[pos];
		x[pos] = t;

		if (pos == k)
			break;
		if (pos < k)
			left = pos + 1;
		else
			right = pos - 1;
	}
	return x[k];
}

__global__ void mf_cuda_sm(unsigned __int64 *padded_array, int padded_array_size, int window_size, unsigned __int64* filtered_array, int original_size, int mid){
	
	__shared__ int smem[BLOCK_WIDTH*BLOCK_HEIGHT];
	
	unsigned int window[441];
	
	int x = blockIdx.x * (BLOCK_WIDTH - (2 * mid)) + threadIdx.x;
	int y = blockIdx.y * (BLOCK_WIDTH - (2 * mid)) + threadIdx.y;

	int x_pad = blockIdx.x * BLOCK_WIDTH + threadIdx.x - blockIdx.x*mid*2 ;
	int y_pad = blockIdx.y * BLOCK_WIDTH + threadIdx.y - blockIdx.y*mid*2 ;

	if (x_pad >= padded_array_size || y_pad >= padded_array_size)
		return;

	unsigned int index = y_pad * padded_array_size + x_pad;
	unsigned int bindex = threadIdx.y * blockDim.x + threadIdx.x;

	smem[bindex] = padded_array[index];
	__syncthreads();

	if ((threadIdx.x >= mid) && (threadIdx.x < (BLOCK_WIDTH - mid)) &&
		(threadIdx.y >= mid) && (threadIdx.y < (BLOCK_HEIGHT - mid)) &&
		((x) < original_size + mid) && ((y) < original_size + mid)){
		int sum = 0;
		for (int dy = -mid; dy <= mid; dy++){
			for (int dx = -mid; dx <= mid; dx++){
				window[sum] = smem[bindex + (dy*blockDim.x) + dx];
				sum++;
				
			}
		}

		/*for (int i = 0; i < (window_size*window_size + 1) / 2; ++i) {
			int minval = i;
			for (int l = i + 1; l < (window_size*window_size); ++l)
				if (window[l] < window[minval])
					minval = l;

			unsigned int temp = window[i];
			window[i] = window[minval];
			window[minval] = temp;
		}*/
		unsigned int median = quick_select(window_size*window_size / 2, window, window_size*window_size);

		filtered_array[(y - mid) * original_size + (x - mid)] = median;//window[(window_size*window_size) / 2];
	}
};

__global__ void mf_cuda(unsigned __int64 *padded_array, int padded_array_size, int window_size, unsigned __int64* filtered_array, int original_size, int mid)
{
	//printf("%d\n", original_size);
	unsigned int window[441];
	int count;

	int x = blockDim.x * blockIdx.x + threadIdx.x + mid;
	int y = blockDim.y * blockIdx.y + threadIdx.y + mid;

	if ((x >= original_size + mid) || (y >= original_size + mid))
		return;

	count = 0;
	for (int c = y - mid; c <= y + mid; c++) {
		for (int r = x - mid; r <= x + mid; r++) {
			window[count] = padded_array[padded_array_size*c + r];
			count++;
		}
	}

	/*for (int i = 0; i < (window_size*window_size + 1) / 2; ++i) {
		int minval = i;
		for (int l = i + 1; l < (window_size*window_size); ++l)
			if (window[l] < window[minval])
				minval = l;

		unsigned int temp = window[i];
		window[i] = window[minval];
		window[minval] = temp;
	}*/

	unsigned int median = quick_select(window_size*window_size / 2, window, window_size*window_size);

	filtered_array[original_size*(y - mid) + (x - mid)] = median;//window[(window_size*window_size) / 2];
};

unsigned __int64 *array_padder(int original_size, int window_size, unsigned __int64 *original_array){
	
	int pad_val = (window_size - 1) / 2;
	int padded_size = (original_size + window_size - 1);

	unsigned __int64* padded_array;
	padded_array = new unsigned __int64[padded_size*padded_size]();

	for (int i = 0; i < padded_size*padded_size; i++){
		padded_array[i] = 0;
	}
	
	for (int i = pad_val; i < padded_size - pad_val; ++i){
		for (int j = pad_val; j < padded_size - pad_val; ++j){
			padded_array[padded_size*i+j] = original_array[original_size*(i - pad_val)+(j - pad_val)];
		}
	}

	//Mirroring
	for (int x = pad_val; x < pad_val + original_size; ++x){
		int count = 0;
		for (int top = pad_val - 1; top >= 0; --top){
			padded_array[padded_size*top+x] = original_array[original_size*count+(x - pad_val)];
			padded_array[padded_size*x + top] = original_array[original_size*(x - pad_val)+count];
			count++;
		}
	}

	for (int y = original_size - 1; y >= 0; --y){
		for (int space = 0; space < pad_val; space++){
			padded_array[padded_size*(y + pad_val)+(padded_size - pad_val + space)] = original_array[original_size*y+(original_size - 1 - space)];
		}
	}

	int tmpVal = original_size - 1;
	for (int y = original_size + pad_val; y < padded_size; ++y){
		for (int x = 0 + pad_val; x < original_size + pad_val; ++x){
			padded_array[padded_size*y+x] = original_array[original_size*tmpVal+(x - pad_val)];
		}
		tmpVal--;
	}
	//End mirroring

	return padded_array;
}

void medianFilter(vector<float> bins, int window_size, unsigned __int64* unfiltered_array, int unfiltered_x, int unfiltered_y){
	int p, mid = (window_size - 1) / 2;

	cout << "::Create window array" << endl;

	unsigned int* window;
	window = new unsigned int[window_size*window_size]();
	for (int i = 0; i < window_size*window_size; i++){
		window[i] = 0;
	}

	cout << "::Create 2D filtered array" << endl;
	unsigned __int64* filtered_points;
	filtered_points = new unsigned __int64[unfiltered_x*unfiltered_x]();
	for (int i = 0; i < unfiltered_x*unfiltered_y; i++){
			filtered_points[i] = 0;
	}

	double startSerialPad = omp_get_wtime();
	unsigned __int64* padded_array = array_padder(unfiltered_x, window_size, unfiltered_array);
	double endSerialPad = omp_get_wtime();
	cout << "::Array padded:\t\t\t" << endSerialPad - startSerialPad << endl;

	cout << "\n==Enter serial loop==" << endl;
	double startSerial = omp_get_wtime();

	int padded_array_size = unfiltered_x + window_size - 1;

	for (int column = mid; column < padded_array_size - mid; column++)
	{
		for (int row = mid; row < padded_array_size - mid; row++)
		{
			p = 0;
			for (int c = column - mid; c <= column + mid; c++)
				for (int r = row - mid; r <= row + mid; r++)
				{
					window[p] = padded_array[padded_array_size*c+r];
					p++;
				}
			/*for (int i = 0; i<(window_size*window_size+1)/2; ++i) {

				int min = i;
				for (int l = i + 1; l<(window_size*window_size); ++l) 
					if (window[l] < window[min])
						min = l;

				unsigned int temp = window[i];
				window[i] = window[min];
				window[min] = temp;
			}*/
			unsigned int median = quick_select(window_size*window_size / 2, window, window_size*window_size);
			//printf("[%d]\n", median);
			filtered_points[unfiltered_x*(column - mid)+(row - mid)] = median;
		}
	} 

	double endSerial = omp_get_wtime();

	cout << "::Loop time serial:\t\t" << endSerial - startSerial << endl;

	/*for (int i = 0; i < unfiltered_x; i++){
		for (int j = 0; j < unfiltered_y; j++){
			cout << filtered_points[unfiltered_x*i + j] << " ";
		}
		cout << endl;
	}*/

	printToFile(bins, filtered_points, unfiltered_x, unfiltered_y, "filtered_serial.csv");

	delete[] filtered_points;
	delete[] padded_array;
	delete[] window;
};

int medianFilter_CUDA(vector<float> bins, int window_size, unsigned __int64* unfiltered_array, int unfiltered_x, int unfiltered_y){

	int mid = (window_size - 1) / 2;

	int TILE_W = BLOCK_WIDTH - 2 * mid;
	int TILE_H = BLOCK_HEIGHT - 2 * mid;

	int padded_arr_size = unfiltered_x + window_size - 1;
	cout << "::Create 2D filtered array" << endl;

	unsigned __int64 *result_array;
	result_array = new unsigned __int64[unfiltered_x*unfiltered_y]();

	unsigned __int64 *d_filtered_array;

	double startSerialPad = omp_get_wtime();
	
	unsigned __int64* padded_array = array_padder(unfiltered_x, window_size, unfiltered_array);
	unsigned __int64 *d_padded_array;
	
	double endSerialPad = omp_get_wtime();
	cout << "::Array padded:\t\t\t" << endSerialPad - startSerialPad << endl;

	cout << "\n==Enter CUDA loop==" << endl;
	
	//FILTERED ARRAY ALLOC
	if (hipMalloc(&d_filtered_array, (unfiltered_x*unfiltered_y)*sizeof(unsigned __int64)) != hipSuccess){
		cout << "Error allocating filtered array space on device" << endl;
		return 0;
	}

	//PADDED ARRAY ALLOC/COPY
	if (hipMalloc(&d_padded_array, (padded_arr_size*padded_arr_size)*sizeof(unsigned __int64)) != hipSuccess){
		cout << "Error allocating padded_array space on device" << endl;
		hipFree(d_filtered_array);
		return 0;
	}
	if (hipMemcpy(d_padded_array, padded_array, (padded_arr_size*padded_arr_size)*sizeof(unsigned __int64), hipMemcpyHostToDevice) != hipSuccess){
		cout << "Error copying window array to GPU" << endl;
		hipFree(d_filtered_array);
		hipFree(d_padded_array);
		return 0;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	const dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
	const dim3 grid((padded_arr_size + TILE_W - 1) / TILE_W, (padded_arr_size + TILE_H - 1) / TILE_H);

	const dim3 grid_no_sm((unfiltered_x + block.x - 1) / block.x, (unfiltered_y + block.y - 1) / block.y );
	
	cout << "Execute CUDA Kernel SM\n";
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(mf_cuda), hipFuncCachePreferL1);
	hipEventRecord(start);
	mf_cuda_sm<<<grid, block>>>(d_padded_array, padded_arr_size, window_size, d_filtered_array, unfiltered_x, mid);
	hipEventRecord(stop);
	hipDeviceSynchronize();

	if (hipMemcpy(result_array, d_filtered_array, (unfiltered_x*unfiltered_y)*sizeof(unsigned __int64), hipMemcpyDeviceToHost) != hipSuccess){
		cout << "Error copying array back from GPU" << endl;
	}

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "::Loop time CUDA:\t\t" << milliseconds/(float)1000 << endl;

	printToFile(bins, result_array, unfiltered_x, unfiltered_y, "filtered_parallel_sm.csv");
	
	hipFree(d_filtered_array);
	hipFree(d_padded_array);

	//delete[] filtered_array;
	//delete[] padded_array;
	delete[] result_array;


	// NO SHARED MEM KERNEL

	result_array = new unsigned __int64[unfiltered_x*unfiltered_y]();

	//FILTERED ARRAY ALLOC
	if (hipMalloc(&d_filtered_array, (unfiltered_x*unfiltered_y)*sizeof(unsigned __int64)) != hipSuccess){
		cout << "Error allocating filtered array space on device" << endl;
		return 0;
	}

	//PADDED ARRAY ALLOC/COPY
	if (hipMalloc(&d_padded_array, (padded_arr_size*padded_arr_size)*sizeof(unsigned __int64)) != hipSuccess){
		cout << "Error allocating padded_array space on device" << endl;
		hipFree(d_filtered_array);
		return 0;
	}
	if (hipMemcpy(d_padded_array, padded_array, (padded_arr_size*padded_arr_size)*sizeof(unsigned __int64), hipMemcpyHostToDevice) != hipSuccess){
		cout << "Error copying window array to GPU" << endl;
		hipFree(d_filtered_array);
		hipFree(d_padded_array);
		return 0;
	}


	start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	cout << "\nExecute CUDA Kernel No SM\n";
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(mf_cuda), hipFuncCachePreferL1);
	hipEventRecord(start);
	mf_cuda << <grid_no_sm, block >> >(d_padded_array, padded_arr_size, window_size, d_filtered_array, unfiltered_x, mid);
	hipEventRecord(stop);
	hipDeviceSynchronize();

	if (hipMemcpy(result_array, d_filtered_array, (unfiltered_x*unfiltered_y)*sizeof(unsigned __int64), hipMemcpyDeviceToHost) != hipSuccess){
		cout << "Error copying array back from GPU" << endl;
	}

	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "::Loop time CUDA:\t\t" << milliseconds / (float)1000 << endl;

	printToFile(bins, result_array, unfiltered_x, unfiltered_y, "filtered_parallel_no_sm.csv");

	hipFree(d_filtered_array);
	hipFree(d_padded_array);


	return 0;
};